
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <ctime>

// Kernel para calcular el producto punto de dos arreglos
__global__ void dotProduct(float *a, float *b, float *result, int N) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    // Cada hilo realiza una parte del trabajo
    while (tid < N) {
        result[tid] += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }
}

int main() {
    // Tamaño del arreglo (debe ser múltiplo de 4 y mayor que 500)
    const int N = 1000;

    // Inicialización de los arreglos en el host
    float *h_a = new float[N];
    float *h_b = new float[N];
    float *h_result = new float[N];

    // Inicializa los arreglos con valores constantes
    for (int i = 0; i < N; ++i) {
        h_a[i] = 0.87654321;
        h_b[i] = 0.12345678;
    }

    // Allocación de memoria en el dispositivo
    float *d_a, *d_b, *d_result;
    hipMalloc((void**)&d_a, N * sizeof(float));
    hipMalloc((void**)&d_b, N * sizeof(float));
    hipMalloc((void**)&d_result, N * sizeof(float));

    // Copia de los datos del host al dispositivo
    hipMemcpy(d_a, h_a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * sizeof(float), hipMemcpyHostToDevice);

    // Configuración del lanzamiento del kernel
    int blockSize = 256;
    int gridSize = (N + blockSize - 1) / blockSize;

    // Medición del tiempo de inicio
    clock_t start_time = clock();

    // Lanzamiento del kernel
    dotProduct<<<gridSize, blockSize>>>(d_a, d_b, d_result, N);

    // Sincronización
    hipDeviceSynchronize();

    // Medición del tiempo de finalización
    clock_t end_time = clock();
    double elapsed_time = static_cast<double>(end_time - start_time) / CLOCKS_PER_SEC;

    // Copia del resultado desde el dispositivo al host
    hipMemcpy(h_result, d_result, N * sizeof(float), hipMemcpyDeviceToHost);

    // Suma de todos los resultados parciales para obtener el resultado total
    float totalResult = 0.0;
    for (int i = 0; i < N; ++i) {
        totalResult += h_result[i];
    }

    // Impresión del resultado total y tiempo de ejecución
    std::cout << "Resultado total del producto punto: " << totalResult << std::endl;
    std::cout << "Tiempo de ejecución: " << elapsed_time << " segundos" << std::endl;

    // Liberación de memoria en el dispositivo
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_result);

    // Liberación de memoria en el host
    delete[] h_a;
    delete[] h_b;
    delete[] h_result;

    return 0;
}
